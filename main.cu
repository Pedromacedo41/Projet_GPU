#include "hip/hip_runtime.h"
/******************************************
   Authors:  Pedro Macedo Flores and Hudson Braga Vieira 
   Projet: Batch merge path sort
   Sorbonne Université - Master 2
   Massive parallel programming on GPU devices for Big Data 
   Paris, mars 2021
*******************************************/

#include <cstddef>
#include <iostream>
#include <iterator>
#include <ostream>
#include <stdio.h>
#include <random>
#include <algorithm>  
#include <vector>
#include "timer.h"
#include <random>
#include "utils.h"
#include <string>

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


// using global memory (gpu) . Not yet optimized, not testes
__device__ void trifusion(int * a, int* b, int * sol, int modA, int modB, int idx){
    int * K = new int[2]; 
    int * P = new int[2]; 
    int * Q = new int[2];

    // initial setup
    bool aux1 = (idx > modA);
    K[0]= P[1] = aux1* (idx-modA);
    K[1]= P[0] = aux1*modA + (1-aux1)*idx;

    bool loop_bool = true;
    while(loop_bool){
        /*********************
          set Q position after K or P move following binary search. 
         ( P move 1 segment bellow Q or K move 1 segment above Q if break condition is not met yet)
        *********************/
        // mid distance between K and P 
        int offset = abs(K[1]-P[1])/2; 
        // midpoint in diagonal
        Q[0]= K[0]+offset;
        Q[1]=  K[1]-offset;

        /********************
          P move one segment bellow Q in schema 1, 1  (bottom left = 1) 
          K move one segment above Q in schema 0, 0 (upper right = 0 )
          break condition: schema  0, 1 
        *********************/
        bool upper_right = !(a[Q[1]-1] > b[Q[0]]);
        bool bottom_left = (a[Q[1]] < b[Q[0]-1]);
        // in break condition, tells if upper left is 0 or 1. 
        bool from_upper_or_left = (a[Q[1]] <= b[Q[0]]);

        P[1] = bottom_left*(Q[1]+1);
        P[0] = bottom_left*(Q[0]-1);
        K[1] = (!bottom_left)*(Q[1]-1);
        K[0] = (!bottom_left)*(Q[0]+1);

        // only really updates in schema 0,1 
        sol[idx]+= upper_right* (!bottom_left) * (from_upper_or_left*a[Q[1]] + (!from_upper_or_left)*b[Q[0]]);
        loop_bool =  upper_right* (!bottom_left);
    }
}

__global__ void trifusion_kernel_test(int * a, int* b, int * sol, int modA, int modB){
    //trifusion(a, b, sol, modA, modB, threadIdx.x);
}



void trifusion_test(void){
    int M = 20;
    int modA= 5 , modB = M-modA;
    int maxAB = (modA > modB)? modA : modB;

    // random sorted vectors
    int * a = rand_int_array_sorted(modA);
    int * b = rand_int_array_sorted(modB);
    int * aGPU, *bGPU, * solGPU, *solCPU = new int[M];

    // memory alloc
    testCUDA(hipMalloc(&aGPU, maxAB*sizeof(int)));
    testCUDA(hipMalloc(&bGPU, maxAB*sizeof(int)));
    testCUDA(hipMalloc(&solGPU, M*sizeof(int)));


    /***********************
         CPU run
    ************************/
    Timer timer;
    timer.start();
    int * sol = merge_sequential(a, b, modA, modB);
    timer.add();

    if(check_solution(sol, a, b, modA, modB)) std::cout << "Sequential solution OK" << std::endl; 
    else std::cout << "Sequential solution Wrong" << std::endl;
    std::cout << "Elapsed CPU time: " << timer.getsum()*1000 << " ms" << std::endl << std::endl;


    /***********************
         GPU run
    ************************/
    testCUDA(hipMemcpy(aGPU,a, modA * sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(bGPU,b, modB * sizeof(int), hipMemcpyHostToDevice));

    // timer block
    float TimeVar;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop)); 
    testCUDA(hipEventRecord(start,0));
    // timer block
    
    // execution block
    trifusion_kernel_test<<<1, M>>>(aGPU, bGPU, solGPU, modA, modB);
    // execution block

    //timer block
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // timer block
    
    testCUDA(hipMemcpy(solCPU, solGPU,  M * sizeof(int), hipMemcpyDeviceToHost));
    
    if(check_solution(solCPU, a, b, modA, modB)) std::cout << "Parallel solution OK" << std::endl; 
    else std::cout << "Parallel solution Wrong" << std::endl;
    std::cout << "Elapsed GPU time: " <<  TimeVar << " ms" << std::endl << std::endl;

    /***********************
     Memory Free
    ***********************/

    testCUDA(hipFree(aGPU));
    testCUDA(hipFree(bGPU));
    testCUDA(hipFree(solGPU));

    // memory free
    delete [] a;
    delete [] b;
    delete [] sol;
    delete [] solCPU;

}


__global__ void kernel_batch_sort(int * M, int i){


}

bool check_solution_batch(int *mCPU, int * mSOL,  int d, int batch_dim){
    f(i, batch_dim){
        f(j, d){
            if(mCPU[i*2*d +j] != mSOL[i*2*d+j + ((int) log(d) -1)%2]) return false;
        }
    }
    return true;
}

void cpu_batch_sort(int *mCPU, int d, int batch_dim){
    Timer timer;
    timer.start();
    f(i, batch_dim){
       std::sort(&mCPU[2*d*i], &mCPU[2*d*i+d]);
    }
    timer.add();

    std::cout << "Elapsed CPU time: " << timer.getsum()*1000 << " ms" << std::endl << std::endl;

}


void batch_sort(int d, int batch_dim, int max_threads_per_block){

    // store on GPU a vectot M of size  2 * batch_dim * d
    // copy each vector j to A[j][0....d]  (setting 0 to  A[j][d+1, ...2d-1]
    // A[batch_id][ 0, ... d//2] keeps old values and A[batch_id][d//2+1, ....d]  new ones or vice versa,  using i%2 trick 
    int * mCPU = rand_int_array(2*d*batch_dim);
    int * mSOL = new int[2*d*batch_dim];
    
    int * mGPU;
    
    testCUDA(hipMalloc(&mGPU,2*d*batch_dim*sizeof(int)));
    testCUDA(hipMemcpy(mGPU,mCPU, 2*d*batch_dim*sizeof(int), hipMemcpyHostToDevice));
    
    cpu_batch_sort(mCPU, d , batch_dim);
    int mul = (d>max_threads_per_block)? (d / max_threads_per_block) : 1;

    // timer block
    float TimeVar;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop)); 
    testCUDA(hipEventRecord(start,0));
    // timer block
    
    // execution block
    f(i, log(d)){
        // for each vector to sort, 2**( log d - i -1) merges to do, each merge take 2**(i+1) threads  => always d threads on total 
	    kernel_batch_sort<<< batch_dim*mul, (d > max_threads_per_block)? max_threads_per_block: d >>> (mGPU, i);
	    hipDeviceSynchronize();
    } 
    // execution block

    //timer block
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // timer block

    testCUDA(hipMemcpy(mSOL, mGPU,  2*d*batch_dim*sizeof(int), hipMemcpyDeviceToHost));

    if(check_solution_batch(mCPU, mSOL, d, batch_dim)) std::cout << "Parallel solution OK" << std::endl; 
    else std::cout << "Parallel solution Wrong" << std::endl;
    std::cout << "Elapsed GPU time: " <<  TimeVar << " ms" << std::endl << std::endl;
    
    // memory free
    testCUDA(hipFree(mGPU));
    delete [] mCPU;
}


int main(int argc, char * argv[]){
    // cin and cout as fast as printf
	std::ios_base::sync_with_stdio(false);
    //trifusion_test();

    int d = 256;
    int batch_dim = 2;

    if(argc==3){
        d = std::stoi(argv[1]);
        batch_dim= std::stoi(argv[2]);
    }

    if(isPowerOfTwo(d)){
        // check the number of SM and the parameters given 
        hipDeviceProp_t prop; 
	    testCUDA(hipGetDeviceProperties(&prop,0));
        std::cout << "GPU informations " << std::endl;
        std::cout << "-----------------" << std::endl;
	    std::cout <<  "Max threads per block: " << prop.major << std::endl;
	    std::cout <<  "SM count: " << prop.multiProcessorCount << std::endl << std::endl;

        int mul = (d>prop.maxThreadsPerBlock)? (d / prop.maxThreadsPerBlock) : 1;
        if(mul*batch_dim > prop.multiProcessorCount){
            std::cout << "WARNING: number of blocks greater than GPU SM count" << std::endl;
        }
    
        batch_sort(d,batch_dim, prop.maxThreadsPerBlock);
    }else{
        std::cout << "ABORTED: d is not power of 2" << std::endl;
    }

    return 0; 
}



