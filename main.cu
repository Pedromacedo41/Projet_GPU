#include "hip/hip_runtime.h"
/******************************************
   Authors:  Pedro Macedo Flores and Hudson Braga Vieira 
   Projet: Batch merge path sort
   Sorbonne Université - Master 2
   Massive parallel programming on GPU devices for Big Data 
   Paris, mars 2021
*******************************************/

#include <cuda_device_runtime_api.h>
#include <iostream>
#include <iterator>
#include <ostream>
#include <stdio.h>
#include <random>
#include <algorithm>  
#include <vector>
#include "timer.h"
#include <random>
#include "utils.h"

#define cl(x, v) memset((x), (v), sizeof(x))
#define f(i, t) for(int (i) = 0; (i) < (t); (i)++)

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d, code %d \n", file, line, error);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


int * merge_sequential(int * a, int* b, int modA, int modB){
    int *sol= new int[modA+modB-1];

    int i=0, j = 0;
    while((i+j)< (modA+modB)){
        if(i>=modA){
            sol[i+j]= b[j];
            j++;

        }else if (j >= modB || a[i] < b[j]) {
            sol[i+j] = a[i];       // goes 
            i++;
        }else {
            sol[i+j]= b[j];        // goes right
            j++;
        }

    }

    return sol;
}


__global__ void kernel_k(int * a, int* b, int * sol, int modA, int modB){
    int idx = threadIdx.x;
}



int main(void){
    // cin and cout as fast as printf
	std::ios_base::sync_with_stdio(false);

    int M = 20;
    int modA= 5 , modB = M-modA;
    int maxAB = (modA > modB)? modA : modB;

    // random sorted vectors
    int * a = rand_int_array(modA);
    int * b = rand_int_array(modB);
    int * aGPU, *bGPU, * solGPU, *solCPU = new int[M];

    // memory alloc
    testCUDA(hipMalloc(&aGPU, maxAB*sizeof(int)));
    testCUDA(hipMalloc(&bGPU, maxAB*sizeof(int)));
    testCUDA(hipMalloc(&solGPU, M*sizeof(int)));


    /***********************
         CPU run
    ************************/
    Timer timer;
    timer.start();
    int * sol = merge_sequential(a, b, modA, modB);
    timer.add();

    if(check_solution(sol, a, b, modA, modB)) std::cout << "Sequential solution OK" << std::endl; 
    else std::cout << "Sequential solution Wrong" << std::endl;
    std::cout << "Elapsed CPU time: " << timer.getsum()*1000 << " ms" << std::endl << std::endl;


    /***********************
         GPU run
    ************************/
    testCUDA(hipMemcpy(aGPU,a, modA * sizeof(int), hipMemcpyHostToDevice));
    testCUDA(hipMemcpy(bGPU,b, modB * sizeof(int), hipMemcpyHostToDevice));

    // timer block
    float TimeVar;
    hipEvent_t start, stop;
    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop)); 
    testCUDA(hipEventRecord(start,0));
    // timer block
    
    // execution block
    kernel_k<<<1, M>>>(aGPU, bGPU,  solGPU, modA, modB);
    // execution block

    //timer block
    testCUDA(hipEventRecord(stop,0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(&TimeVar, start, stop));
    // timer block
    
    testCUDA(hipMemcpy(solCPU, solGPU,  M * sizeof(int), hipMemcpyDeviceToHost));
    
    if(check_solution(solCPU, a, b, modA, modB)) std::cout << "Parallel solution OK" << std::endl; 
    else std::cout << "Parallel solution Wrong" << std::endl;
    std::cout << "Elapsed GPU time: " <<  TimeVar << " ms" << std::endl << std::endl;

    /***********************
     Memory Free
    ***********************/

    testCUDA(hipFree(aGPU));
    testCUDA(hipFree(bGPU));
    testCUDA(hipFree(solGPU));

    // memory free
    delete [] a;
    delete [] b;
    delete [] sol;
    delete [] solCPU;

    return 0; 
}



